#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

using namespace std;

__global__ void findEachMin(double* v1, double* v2, double* vr, int sz)
{
	int i = threadIdx.x;
	vr[i] = v1[i] < v2[i] ? v1[i] : v2[i];
}

hipError_t findEachMinCuda(double* h_v1, double* h_v2, double* h_vr, int sz)
{
	double* d_v1;
	double* d_v2;
	double* d_vr;
	hipError_t error;
	if (!
		(
			(error = hipMalloc((void**)&d_v1, sizeof(double) * sz))
			||
			(error = hipMalloc((void**)&d_v2, sizeof(double) * sz))
			||
			(error = hipMalloc((void**)&d_vr, sizeof(double) * sz))
			||
			(error = hipMemcpy(d_v1, h_v1, sizeof(double) * sz, hipMemcpyHostToDevice))
			||
			(error = hipMemcpy(d_v2, h_v2, sizeof(double) * sz, hipMemcpyHostToDevice))
			||
			(error = hipMemcpy(d_v2, h_v2, sizeof(double) * sz, hipMemcpyHostToDevice))
			)
		)
	{
		findEachMin << <1, sz >> > (d_v1, d_v2, d_vr, sz);
		if (!(error = hipGetLastError()))
		{
			error = hipMemcpy(h_vr, d_vr, sizeof(double) * sz, hipMemcpyDeviceToHost);
		}
	}

	hipFree(d_v1);
	hipFree(d_v2);
	hipFree(d_vr);
	return error;
}

int main()
{
	int res = 0;
	int sz;
	double* h_v1;
	double* h_v2;
	double* h_vr;
	cin >> sz;
	h_v1 = new double[sz];
	h_v2 = new double[sz];
	h_vr = new double[sz];
	int j;

	for (int i = 0; i < sz; i++)
	{
		cin >> h_v1[i];
	}

	for (int i = 0; i < sz; i++)
	{
		cin >> h_v2[i];
	}
	hipError_t error;
	if ((error = findEachMinCuda(h_v1, h_v2, h_vr, sz)) != hipSuccess)
	{
		cerr << "Execution CUDA error: " << error << endl;
		res = 1;
	}
	else
	{
		cout << scientific;
		cout.precision(10);
		for (int i = 0; i < sz; i++)
		{
			cout << h_vr[i];
			if (i != sz - 1)
			{
				cout << " ";
			}
		}
	}

	delete[] h_v1;
	delete[] h_v2;
	delete[] h_vr;
	return res;
}
